#include "hip/hip_runtime.h"
/* 
 * Copyright 2015-2016 NVIDIA Corporation. All rights reserved.
 *
 * Sample to demonstrate use of NVlink CUPTI APIs
 * 
 * This version is significantly changed to use PAPI and the CUDA component to
 * handle access and reporting. As of 10/05/2018, I have deleted all CUPTI_ONLY
 * references, for clarity. The file nvlink_bandwidth_cupti_only.cu contains
 * the cupti-only code.  I also deleted the #if PAPI; there is no option
 * without PAPI.  Also, before my changes, the makefile did not even have a
 * build option that set CUPTI_ONLY for this file.
 *
 * -TonyC. 
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include "papi.h"

// THIS MACRO EXITS if the papi call does not return PAPI_OK. Do not use for routines that
// return anything else; e.g. PAPI_num_components, PAPI_get_component_info, PAPI_library_init.
#define CALL_PAPI_OK(papi_routine)                                                        \
    do {                                                                                  \
        int _papiret = papi_routine;                                                      \
        if (_papiret != PAPI_OK) {                                                        \
            fprintf(stderr, "%s:%d: PAPI Error: function %s failed with ret=%d [%s].\n",  \
                    __FILE__, __LINE__, #papi_routine, _papiret, PAPI_strerror(_papiret));\
            exit(-1);                                                                     \
        }                                                                                 \
    } while (0);


#define CUPTI_CALL(call)                                                \
    do {                                                                \
        CUptiResult _status = call;                                     \
        if (_status != CUPTI_SUCCESS) {                                 \
            const char *errstr;                                         \
            cuptiGetResultString(_status, &errstr);                     \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
                    __FILE__, __LINE__, #call, errstr);                 \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define DRIVER_API_CALL(apiFuncCall)                                    \
    do {                                                                \
        hipError_t _status = apiFuncCall;                                 \
        if (_status != hipSuccess) {                                  \
            const char *errName=NULL, *errStr=NULL;                     \
            hipError_t _e1 = hipDrvGetErrorName(_status, &errName);           \
            hipError_t _e2 = hipDrvGetErrorString(_status, &errStr);          \
            fprintf(stderr, "%s:%d: error: function %s failed with error %d [%s]='%s'.\n", \
                    __FILE__, __LINE__, #apiFuncCall, _status, errName, errStr);           \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define RUNTIME_API_CALL(apiFuncCall)                                   \
    do {                                                                \
        hipError_t _status = apiFuncCall;                              \
        if (_status != hipSuccess) {                                   \
            fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
                    __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status)); \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  

#define MEMORY_ALLOCATION_CALL(var)                                     \
    do {                                                                \
        if (var == NULL) {                                              \
            fprintf(stderr, "%s:%d: Error: Memory Allocation Failed \n", \
                    __FILE__, __LINE__);                                \
            exit(-1);                                                   \
        }                                                               \
    } while (0);  


#define MAX_DEVICES    (32)
#define BLOCK_SIZE     (1024)
#define GRID_SIZE      (512)
#define BUF_SIZE       (32 * 1024)
#define ALIGN_SIZE     (8)
#define SUCCESS        (0)
#define MAX_SIZE       (64*1024*1024)   // 64 MB

int Streams;                            // Number of physical copy engines to use; taken from Device Properties asyncEngineCount.
int cpuToGpu = 0;
int gpuToGpu = 0;


//-----------------------------------------------------------------------------
// This is the GPU routine to move a block from dst (on one GPU) to src (on
// another GPU. This is no longer used in this code; we use hipMemcpyAsync().
// Typical invocation (depends on #defines above):
//
//  for(i = 0; i < Streams; i++) {
//      test_nvlink_bandwidth <<< GRID_SIZE, BLOCK_SIZE >>> ((float *) pDevBuffer1[i], (float *) pDevBuffer0[i]);
//      printf("test_nvlink_bandwidth stream %d \n", i);
//  }
//-----------------------------------------------------------------------------
extern "C" __global__ void test_nvlink_bandwidth(float *src, float *dst)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx % 2) dst[idx] = src[idx] * 2.0f;
    else         dst[idx] = src[idx] * 1.5f;
//  dst[idx] = src[idx] * 2.0f;
} // end routine

#define DIM(x) (sizeof(x)/sizeof(*(x)))


//-----------------------------------------------------------------------------
// Return a text version with B, KB, MB, GB or TB. 
//-----------------------------------------------------------------------------
void calculateSize(char *result, uint64_t size)
{
    int i;

    const char *sizes[] = { "TB", "GB", "MB", "KB", "B" };
    uint64_t exbibytes = 1024ULL * 1024ULL * 1024ULL * 1024ULL;

    uint64_t multiplier = exbibytes;

    for(i = 0; (unsigned) i < DIM(sizes); i++, multiplier /= (uint64_t) 1024) {
        if(size < multiplier)
            continue;
        sprintf(result, "%.1f %s", (float) size / multiplier, sizes[i]);
        return;
    }
    strcpy(result, "0");
    return;
} // end routine


//-----------------------------------------------------------------------------
// We use Async copies (returns while operation is still in progress) with 
// multiple streams; hipDeviceSynchronize waits for them to complete.
//-----------------------------------------------------------------------------
void testCpuToGpu(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer, float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;
    fprintf(stderr, "Streams = %d.\n", Streams); 
    // Unidirectional copy H2D (Host to Device).
    for(i = 0; i < Streams; i++) {
//      RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
        test_nvlink_bandwidth <<< GRID_SIZE, BLOCK_SIZE >>> ((float *) pDevBuffer[i], (float *) pHostBuffer[i]);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Unidirectional copy D2H (Device to Host).
    for(i = 0; i < Streams; i++) {
//      RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i], (void *) pDevBuffer[i], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i]));
        test_nvlink_bandwidth <<< GRID_SIZE, BLOCK_SIZE >>> ((float *) pHostBuffer[i], (float *) pDevBuffer[i]);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Bidirectional copy
    for(i = 0; i < Streams; i += 2) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
        RUNTIME_API_CALL(hipMemcpyAsync(pHostBuffer[i + 1], (void *) pDevBuffer[i + 1], bufferSize, hipMemcpyDeviceToHost, cudaStreams[i + 1]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());
} // end routine.


//-----------------------------------------------------------------------------
// Copy buffers from the host to each device, in preparation for a transfer
// between devices.
// We use Async copies (returns while operation is still in progress) with 
// multiple streams; hipDeviceSynchronize waits for them to complete.
//-----------------------------------------------------------------------------
void testGpuToGpu_part1(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1, 
      float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;

    RUNTIME_API_CALL(hipSetDevice(0));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(1, 0));
    RUNTIME_API_CALL(hipSetDevice(1));
    RUNTIME_API_CALL(hipDeviceEnablePeerAccess(0, 0));

    // Unidirectional copy H2D
    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer0[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }

    RUNTIME_API_CALL(hipDeviceSynchronize());

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer1[i], pHostBuffer[i], bufferSize, hipMemcpyHostToDevice, cudaStreams[i]));
    }

    RUNTIME_API_CALL(hipDeviceSynchronize());
} // end routine.


//-----------------------------------------------------------------------------
// Copy from device zero to device 1, then from device 1 to device 0.
//-----------------------------------------------------------------------------
void testGpuToGpu_part2(CUpti_EventGroup * eventGroup, 
      hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1, 
      float **pHostBuffer, size_t bufferSize, 
      hipStream_t * cudaStreams, uint64_t * timeDuration, 
      int numEventGroup)
{
    int i;

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer0[i], (void *) pDevBuffer1[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
        printf("Copy %zu stream %d to devBuffer0 from devBuffer1 \n", bufferSize, i);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMemcpyAsync((void *) pDevBuffer1[i], (void *) pDevBuffer0[i], bufferSize, hipMemcpyDeviceToDevice, cudaStreams[i]));
        printf("Copy %zu stream %d to devBuffer1 from devBuffer0 \n", bufferSize, i);
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());
} // end routine.


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
static void printUsage()
{
    printf("usage: Demonstrate use of NVlink CUPTI APIs\n");
    printf("       -help           : display help message\n");
    printf("       --cpu-to-gpu    : Show results for data transfer between CPU and GPU \n");
    printf("       --gpu-to-gpu    : Show results for data transfer between two GPUs \n");
} // end routine.


//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
//-----------------------------------------------------------------------------
void parseCommandLineArgs(int argc, char *argv[])
{
    if(argc != 2) {
        printf("Invalid number of options\n");
        exit(0);
    }

    if(strcmp(argv[1], "--cpu-to-gpu") == 0) {
        cpuToGpu = 1;
    } else if(strcmp(argv[1], "--gpu-to-gpu") == 0) {
        gpuToGpu = 1;
    } else if((strcmp(argv[1], "--help") == 0) || 
              (strcmp(argv[1], "-help") == 0)  || 
              (strcmp(argv[1], "-h") == 0)) {
        printUsage();
        exit(0);
    } else {
        cpuToGpu = 1;
    }
} // end routine.


//-----------------------------------------------------------------------------
// Main program.
//-----------------------------------------------------------------------------
int main(int argc, char *argv[])
{
    int deviceCount = 0, i = 0, numEventGroup = 0, eventsRead=0;
    size_t bufferSize = 0, freeMemory = 0, totalMemory = 0;
    char str[64];

    hipDeviceptr_t *pDevBuffer0  = NULL;
    hipDeviceptr_t *pDevBuffer1  = NULL;
    float       **pHostBuffer = NULL;

    hipStream_t *cudaStreams = NULL;
    hipDeviceProp_t prop[MAX_DEVICES];
    uint64_t timeDuration;
    CUpti_EventGroup eventGroup[32];

    // This contains the original set of 'bandwidth' events to read.  This
    // program attempts to read them together within a single EventSet. PAPI
    // allows them all, and the component does not complain, but it always
    // returns 0 for the two "receive" events below. That is not the result
    // when we read each of these separately for the same program; which is the
    // approach taken in the program nvlink_all.cu. I suspect nvlink has a
    // conflict of some sort, but haven't tracked down the documentation to
    // prove that.  -Tony C.

#define NUM_METRIC     ( 4)
    const char *MetricBase[NUM_METRIC] = {
        "cuda:::metric:nvlink_total_data_transmitted"        , // okay Group NVLINK.
        "cuda:::metric:nvlink_transmit_throughput"           , // okay Group NVLINK.
        "cuda:::metric:nvlink_total_data_received"           , // okay Group NVLINK.
        "cuda:::metric:nvlink_receive_throughput"            , // okay Group NVLINK.
//      "cuda:::metric:inst_per_warp"                        , // okay group A.
//      "cuda:::metric:warp_execution_efficiency"            , // okay Group A.
//      "cuda:::metric:warp_nonpred_execution_efficiency"    , // okay Group A.
//      "cuda:::metric:shared_load_transactions_per_request" , // okay Group A.
//      "cuda:::metric:shared_store_transactions_per_request", // okay Group A.
//      "cuda:::metric:shared_store_transactions"            , // okay Group A.
//      "cuda:::metric:shared_load_transactions"             , // okay Group A.
//      "cuda:::metric:inst_replay_overhead"                 , // Group B
//      "cuda:::metric:local_load_transactions"              , // Group B.
//      "cuda:::metric:local_load_transactions_per_request"  , // Group NONE. Bad Combo, even by itself requires 2 passes.
//      "cuda:::metric:local_store_transactions_per_request" , // Group NONE. Bad Combo, even by itself.
//      "cuda:::metric:gld_transactions_per_request"         , // Group NONE. Bad Combo, even by itself.
//      "cuda:::metric:gst_transactions_per_request"         , // Group NONE. Bad Combo, even by itself.
//      "cuda:::event:active_cycles"                         ,
//      "cuda:::event:active_warps"                          ,
//      "cuda:::event:active_cycles"                         ,
//      "cuda:::event:active_warps"                          ,
//      "cuda:::event:inst_executed"                         ,
//      "cuda:::event:warps_launched"                        ,
//      "cuda:::metric:branch_efficiency"                    , // Even by itself, causes signal 11 (seg fault) on SECOND read.
    };
    
    // Parse command line arguments
    parseCommandLineArgs(argc, argv);
    if (cpuToGpu) printf("TEST: CPU to GPU transfer.\n");
    else          printf("TEST: GPU to GPU transfer.\n");

    DRIVER_API_CALL(hipInit(0));
    RUNTIME_API_CALL(hipGetDeviceCount(&deviceCount));
    printf("There are %d devices.\n", deviceCount);

    if(deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(-1);
    }

    Streams = 1;                                            // Always use at least ONE stream.
    for(i = 0; i < deviceCount; i++) {
        RUNTIME_API_CALL(hipGetDeviceProperties(&prop[i], i));
        printf("CUDA Device %d Name: %s", i, prop[i].name);
        printf(", AsyncEngineCount=%i", prop[i].asyncEngineCount);
        printf(", MultiProcessors=%i", prop[i].multiProcessorCount);
        printf(", MaxThreadsPerMP=%i", prop[i].maxThreadsPerMultiProcessor);
        printf("\n");
        if (prop[i].asyncEngineCount > Streams) {           // If a new high,
            Streams = prop[i].asyncEngineCount;             // Always use the maximum.
        }
    }

    printf("Streams to use: %i (= max Copy Engines).\n", Streams);

    // allocate space
    pDevBuffer0 = (hipDeviceptr_t*)    calloc(Streams, sizeof(hipDeviceptr_t));
    pDevBuffer1 = (hipDeviceptr_t*)    calloc(Streams, sizeof(hipDeviceptr_t));
    pHostBuffer = (float **)        calloc(Streams, sizeof(float*));
    cudaStreams = (hipStream_t*)   calloc(Streams, sizeof(hipStream_t));

    // Set memcpy size based on available device memory
    RUNTIME_API_CALL(hipMemGetInfo(&freeMemory, &totalMemory));
    printf("Total Device Memory available : ");
    calculateSize(str, (uint64_t) totalMemory);
    printf("%s\n", str);

    bufferSize = MAX_SIZE < (freeMemory / 4) ? MAX_SIZE : (freeMemory / 4);
    bufferSize = bufferSize/2;
    printf("Memcpy size is set to %llu B (%llu MB)\n", (unsigned long long) bufferSize, (unsigned long long) bufferSize / (1024 * 1024));

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipStreamCreate(&cudaStreams[i]));
    }
    RUNTIME_API_CALL(hipDeviceSynchronize());

    // Nvlink-topology Records are generated even before hipMemcpy API is called.
    CUPTI_CALL(cuptiActivityFlushAll(0x7fffffff)); // flag covers every kind of record.

    fprintf(stderr, "Setup PAPI counters internally (PAPI)\n");
    int EventSet = PAPI_NULL;
    long long values[MAX_DEVICES * NUM_METRIC];
    char *EventName[MAX_DEVICES * NUM_METRIC];
    int eventCount;
    int retval, ee;
    int k, cid=-1;

    /* PAPI Initialization */
    retval = PAPI_library_init(PAPI_VER_CURRENT);
    if(retval != PAPI_VER_CURRENT) {
        fprintf(stderr, "PAPI_library_init failed, ret=%i [%s]\n", 
            retval, PAPI_strerror(retval));
        exit(-1);
    }

    fprintf(stderr, "PAPI version: %d.%d.%d\n", 
        PAPI_VERSION_MAJOR(PAPI_VERSION), 
        PAPI_VERSION_MINOR(PAPI_VERSION), 
        PAPI_VERSION_REVISION(PAPI_VERSION));

    // Find cuda component index.
    k = PAPI_num_components();                                          // get number of components.
    for (i=0; i<k && cid<0; i++) {                                      // while not found,
        PAPI_component_info_t *aComponent = 
            (PAPI_component_info_t*) PAPI_get_component_info(i);        // get the component info.     
        if (aComponent == NULL) {                                       // if we failed,
            fprintf(stderr,  "PAPI_get_component_info(%i) failed, "
                "returned NULL. %i components reported.\n", i,k);
            exit(-1);    
        }

       if (strcmp("cuda", aComponent->name) == 0) cid=i;                // If we found our match, record it.
    } // end search components.

    if (cid < 0) {                                                      // if no PCP component found,
        fprintf(stderr, "Failed to find pcp component among %i "
            "reported components.\n", k);
        exit(-1); 
    }

    fprintf(stderr, "Found CUDA Component at id %d\n",cid);

    CALL_PAPI_OK(PAPI_create_eventset(&EventSet)); 
    CALL_PAPI_OK(PAPI_assign_eventset_component(EventSet, cid)); 

    // ===== Allocate Memory =====================================

    for(i = 0; i < Streams; i++) {
        RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer0[i], bufferSize));

        pHostBuffer[i] = (float *) malloc(bufferSize);
        MEMORY_ALLOCATION_CALL(pHostBuffer[i]);
    }
    
    // Add events at a GPU specific level ... eg cuda:::metric:nvlink_total_data_transmitted:device=0
    char tmpEventName[1024];
    eventCount = 0;
    for(i = 0; i < deviceCount; i++) {                                  // Profile all devices.
        fprintf(stderr, "Set device to %d\n", i);
        for(ee = 0; ee < NUM_METRIC; ee++) {
            snprintf(tmpEventName, 1024, "%s:device=%d\0", MetricBase[ee], i);
            retval = PAPI_add_named_event(EventSet, tmpEventName);      // Don't want to fail program if name not found...
            if(retval == PAPI_OK) {
                EventName[eventCount] = strdup(tmpEventName);
                eventCount++;
            } else {
                fprintf(stderr, "Failed to add event %s to GPU %i; ret=%d [%s].\n", tmpEventName, i, retval, PAPI_strerror(retval));
            }
        }
    }

    if (eventCount > 0) {                                              // If we have events...
        for(i = 0; i < eventCount; i++) values[i] = -1;                // init.

        if(cpuToGpu) {
            RUNTIME_API_CALL(hipSetDevice(1));
            for(i = 0; i < Streams; i++) 
                RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer1[i], bufferSize));
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            RUNTIME_API_CALL(hipSetDevice(0));
            testCpuToGpu(eventGroup, pDevBuffer0, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            RUNTIME_API_CALL(hipSetDevice(1));
            testCpuToGpu(eventGroup, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            RUNTIME_API_CALL(hipSetDevice(0));
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        } else if(gpuToGpu) {
            RUNTIME_API_CALL(hipSetDevice(1));
            for(i = 0; i < Streams; i++) 
                RUNTIME_API_CALL(hipMalloc((void **) &pDevBuffer1[i], bufferSize));

            //  Prepare the copy, load up buffers on each device from the host.
            testGpuToGpu_part1(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);

            // Copy from device 0->1, then device 1->0.
            CALL_PAPI_OK(PAPI_start(EventSet));                             // Start event counters.
            testGpuToGpu_part2(eventGroup, pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize, cudaStreams, &timeDuration, numEventGroup);
            CALL_PAPI_OK(PAPI_stop(EventSet, values));                      // Stop and read values.
        }

        // report each event counted.
        for(i = 0; i < eventCount; i++) {
            if (values[i] >= 0) {                                           // If not still -1,
                eventsRead++;                                               // .. count and report.
                calculateSize(str, (uint64_t) values[i] );
                printf("PAPI %64s: %s \n", EventName[i], str);
            }
        }
    }

    // Program cleanup.
    CALL_PAPI_OK(PAPI_cleanup_eventset(EventSet));                      // Delete all events in set.
    CALL_PAPI_OK(PAPI_destroy_eventset(&EventSet));                     // Release PAPI memory.
    PAPI_shutdown();                                                    // Has no return.
        
    if (eventsRead > 0) {                                               // If we succeeded with any, report. 
        printf("%i bandwidth events successfully reported.\n", eventsRead);
        return(0);                                                      // exit OK.
    }

    printf("Failed to read any bandwidth events.\n");                   // report a failure.
        
    return (-1);                                                        // Exit with error.
} // end MAIN.
