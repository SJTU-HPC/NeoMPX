#include "hip/hip_runtime.h"
/* 
 * PAPI Multiple GPU example.  This example is taken from the NVIDIA
 * documentation (Copyright 1993-2013 NVIDIA Corporation) and has been
 * adapted to show the use of CUPTI and PAPI in collecting event
 * counters for multiple GPU contexts.  PAPI Team (2015)
 */

/*
 * This software contains source code provided by NVIDIA Corporation
 *
 * According to the Nvidia EULA (compute 5.5 version)
 * http://developer.download.nvidia.com/compute/cuda/5_5/rel/docs/EULA.pdf
 *
 * Chapter 2. NVIDIA CORPORATION CUDA SAMPLES END USER LICENSE AGREEMENT
 * 2.1.1. Source Code
 * Developer shall have the right to modify and create derivative works with the Source
 * Code. Developer shall own any derivative works ("Derivatives") it creates to the Source
 * Code, provided that Developer uses the Materials in accordance with the terms and
 * conditions of this Agreement. Developer may distribute the Derivatives, provided that
 * all NVIDIA copyright notices and trademarks are propagated and used properly and
 * the Derivatives include the following statement: “This software contains source code
 * provided by NVIDIA Corporation.”
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cupti.h>
#include <timer.h>

#include "papi.h"
#include "papi_test.h"

#if not defined PAPI
#undef PAPI
#endif

#if not defined CUPTI_ONLY
#undef CUPTI_ONLY
#endif

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#include "simpleMultiGPU.h"

// THIS MACRO EXITS if the papi call does not return PAPI_OK. Do not use for routines that
// return anything else; e.g. PAPI_num_components, PAPI_get_component_info, PAPI_library_init.
#define CALL_PAPI_OK(papi_routine)                                                        \
    do {                                                                                  \
        int _papiret = papi_routine;                                                      \
        if (_papiret != PAPI_OK) {                                                        \
            fprintf(stderr, "%s:%d macro: PAPI Error: function " #papi_routine " failed with ret=%d [%s].\n", \
                    __FILE__, __LINE__, _papiret, PAPI_strerror(_papiret));               \
            exit(-1);                                                                     \
        }                                                                                 \
    } while (0);
// //////////////////////////////////////////////////////////////////////////////
// Data configuration
// //////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
const int DATA_N = 48576 * 32;
#ifdef PAPI
const int MAX_NUM_EVENTS = 32;
#endif

#define CHECK_CU_ERROR(err, cufunc)                                     \
    if (err != hipSuccess) { printf ("Error %d for CUDA Driver API function '%s'\n", err, cufunc); return -1; }

#define CHECK_CUDA_ERROR(err)                                           \
    if (err != hipSuccess) { printf ("%s:%i Error %d for CUDA [%s]\n", __FILE__, __LINE__, err, hipGetErrorString(err) ); return -1; }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                               \
    if (err != CUPTI_SUCCESS) { const char *errStr; cuptiGetResultString(err, &errStr); \
       printf ("%s:%i Error %d [%s] for CUPTI API function '%s'\n", __FILE__, __LINE__, err, errStr, cuptifunc); return -1; }


// //////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
// //////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel( float *d_Result, float *d_Input, int N )
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    
    for( int pos = tid; pos < N; pos += threadN )
        sum += d_Input[pos];
    
    d_Result[tid] = sum;
}

// //////////////////////////////////////////////////////////////////////////////
// Program main
// //////////////////////////////////////////////////////////////////////////////
int main( int argc, char **argv )
{
    // Solver config
    TGPUplan plan[MAX_GPU_COUNT];
    // GPU reduction results
    float h_SumGPU[MAX_GPU_COUNT];
    float sumGPU;
    double sumCPU, diff;
    int i, j, gpuBase, GPU_N;
    
    const int BLOCK_N = 32;
    const int THREAD_N = 256;
    const int ACCUM_N = BLOCK_N * THREAD_N;

    hipCtx_t ctx[MAX_GPU_COUNT];
    
    printf( "Starting simpleMultiGPU\n" );
    
    // Report on the available CUDA devices
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;
    int runtimeVersion = 0, driverVersion = 0;
    char deviceName[64];
    hipDevice_t device[MAX_GPU_COUNT];
    CHECK_CUDA_ERROR( hipGetDeviceCount( &GPU_N ) );
    if( GPU_N > MAX_GPU_COUNT ) GPU_N = MAX_GPU_COUNT;
    printf( "CUDA-capable device count: %i\n", GPU_N );
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CU_ERROR( hipDeviceGet( &device[i], i ), "hipDeviceGet" );
        CHECK_CU_ERROR( hipDeviceGetName( deviceName, 64, device[i] ), "hipDeviceGetName" );
        CHECK_CU_ERROR( hipDeviceGetAttribute( &computeCapabilityMajor, 
            hipDeviceAttributeComputeCapabilityMajor, device[i]), "hipDeviceGetAttribute");
        CHECK_CU_ERROR( hipDeviceGetAttribute( &computeCapabilityMinor, 
            hipDeviceAttributeComputeCapabilityMinor, device[i]), "hipDeviceGetAttribute");
        hipRuntimeGetVersion( &runtimeVersion );
        hipDriverGetVersion( &driverVersion );
        printf( "CUDA Device %d: %s : computeCapability %d.%d runtimeVersion %d.%d driverVersion %d.%d\n", i, deviceName, computeCapabilityMajor, computeCapabilityMinor, runtimeVersion/1000, (runtimeVersion%100)/10, driverVersion/1000, (driverVersion%100)/10 );
        if ( computeCapabilityMajor < 2 ) {
            printf( "CUDA Device %d compute capability is too low... will not add any more GPUs\n", i );
            GPU_N = i;
            break;
        }
    }
    uint32_t cupti_linked_version;
    cuptiGetVersion( &cupti_linked_version );
    printf("CUPTI version: Compiled against version %d; Linked against version %d\n", CUPTI_API_VERSION, cupti_linked_version );
    
    // create one context per device
    for (i = 0; i < GPU_N; i++) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR( hipCtxCreate( &(ctx[i]), 0, device[i] ), "hipCtxCreate" );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    printf( "Generating input data...\n" );
    
    // Subdividing input data across GPUs
    // Get data sizes for each GPU
    for( i = 0; i < GPU_N; i++ )
        plan[i].dataN = DATA_N / GPU_N;
    // Take into account "odd" data sizes
    for( i = 0; i < DATA_N % GPU_N; i++ )
        plan[i].dataN++;
    
    // Assign data ranges to GPUs
    gpuBase = 0;
    for( i = 0; i < GPU_N; i++ ) {
        plan[i].h_Sum = h_SumGPU + i; // point within h_SumGPU array
        gpuBase += plan[i].dataN;
    }

  
    // Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CUDA_ERROR( hipStreamCreate( &plan[i].stream ) );
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Data, plan[i].dataN * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Sum, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Sum_from_device, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Data, plan[i].dataN * sizeof( float ) ) );
        for( j = 0; j < plan[i].dataN; j++ ) {
            plan[i].h_Data[j] = ( float ) rand() / ( float ) RAND_MAX;
        }
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    
    
#ifdef CUPTI_ONLY
//  char const *cuptiEventName = "elapsed_cycles_sm"; // "elapsed_cycles_sm" "inst_executed"; "inst_issued0";
//  char const *cuptiEventName = "inst_executed";     // "elapsed_cycles_sm" "inst_executed"; "inst_issued0";
    char const *cuptiEventName = "inst_per_warp";     // "elapsed_cycles_sm" "inst_executed"; "inst_issued0";
    printf("Setup CUPTI counters internally for event '%s' (CUPTI_ONLY)\n", cuptiEventName);
    CUpti_EventGroup eg[MAX_GPU_COUNT];
    CUpti_EventID *myevent = (CUpti_EventID*) calloc(GPU_N, sizeof(CUpti_EventID));   // Make space for event ids.
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CUPTI_ERROR(cuptiSetEventCollectionMode(ctx[i], CUPTI_EVENT_COLLECTION_MODE_KERNEL), "cuptiSetEventCollectionMode" );
        CHECK_CUPTI_ERROR( cuptiEventGroupCreate( ctx[i], &eg[i], 0 ), "cuptiEventGroupCreate" );
        cuptiEventGetIdFromName ( device[i], cuptiEventName, &myevent[i] );
        printf("GPU %i %s=%u.\n", i, cuptiEventName, myevent[i]);
        CHECK_CUPTI_ERROR( cuptiEventGroupAddEvent( eg[i], myevent[i] ), "cuptiEventGroupAddEvent" );
        CHECK_CUPTI_ERROR( cuptiEventGroupEnable( eg[i] ), "cuptiEventGroupEnable" );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
#endif
    
#ifdef PAPI
    printf("Setup PAPI counters internally (PAPI)\n");
    int EventSet = PAPI_NULL;
    int NUM_EVENTS = MAX_GPU_COUNT*MAX_NUM_EVENTS;
    long long values[NUM_EVENTS];
    int eventCount;
    int cid=-1;
    int retval, ee;
    
    /* PAPI Initialization */
    retval = PAPI_library_init( PAPI_VER_CURRENT );
    if( retval != PAPI_VER_CURRENT ) fprintf( stderr, "PAPI_library_init failed\n" );
    printf( "PAPI version: %d.%d.%d\n", PAPI_VERSION_MAJOR( PAPI_VERSION ), PAPI_VERSION_MINOR( PAPI_VERSION ), PAPI_VERSION_REVISION( PAPI_VERSION ) );
    
    // Find cuda component index.
    int k = PAPI_num_components();                                      // get number of components.
    for (i=0; i<k && cid<0; i++) {                                      // while not found,
        PAPI_component_info_t *aComponent = 
            (PAPI_component_info_t*) PAPI_get_component_info(i);        // get the component info.     
        if (aComponent == NULL) {                                       // if we failed,
            fprintf(stderr,  "PAPI_get_component_info(%i) failed, "
                "returned NULL. %i components reported.\n", i,k);
            exit(-1);    
        }

       if (strcmp("cuda", aComponent->name) == 0) cid=i;                // If we found our match, record it.
    } // end search components.

    if (cid < 0) {                                                      // if no PCP component found,
        fprintf(stderr, "Failed to find cuda component among %i "
            "reported components.\n", k);
        PAPI_shutdown();
        exit(-1); 
    }

    printf("Found CUDA Component at id %d\n", cid);

    CALL_PAPI_OK(PAPI_create_eventset(&EventSet)); 
    CALL_PAPI_OK(PAPI_assign_eventset_component(EventSet, cid)); 
    
    // In this example measure events from each GPU
    int numEventEndings = 2;
    char const *EventEndings[] = { 
        "cuda:::metric:nvlink_total_data_transmitted",
        "cuda:::metric:nvlink_total_data_received",
    };

    // Add events at a GPU specific level ... eg cuda:::device:2:elapsed_cycles_sm
    char *EventName[NUM_EVENTS];
    char tmpEventName[64];
    eventCount = 0;
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );         // Set device
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CUPTI_ERROR(cuptiSetEventCollectionMode(ctx[i], CUPTI_EVENT_COLLECTION_MODE_KERNEL), "cuptiSetEventCollectionMode" );
        for ( ee=0; ee<numEventEndings; ee++ ) {
            snprintf( tmpEventName, 64, "%s:device=%d\0", EventEndings[ee], i );
            // printf( "Trying to add event %s to GPU %d in PAPI...", tmpEventName , i ); fflush(NULL);
            retval = PAPI_add_named_event( EventSet, tmpEventName );
            if (retval==PAPI_OK) {
                printf( "Add event success: '%s' GPU %i\n", tmpEventName, i );
                EventName[eventCount] = (char *)calloc( 64, sizeof(char) );
                snprintf( EventName[eventCount], 64, "%s", tmpEventName );
                eventCount++;
            } else {
                printf( "Add event failure: '%s' GPU %i error=%s\n", tmpEventName, i, PAPI_strerror(retval));
            }
        }
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    
    // Start PAPI event measurement
    retval = PAPI_start( EventSet );
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_start failed, retval=%i [%s].\n", retval, PAPI_strerror(retval));
#endif
    
    // Start timing and compute on GPU(s)
    printf( "Computing with %d GPUs...\n", GPU_N );
    StartTimer();
    
    // Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++) {
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ));
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        // Copy input data from CPU
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof( float ), hipMemcpyHostToDevice, plan[i].stream ) );
        // Perform GPU computations
        reduceKernel <<< BLOCK_N, THREAD_N, 0, plan[i].stream >>> ( plan[i].d_Sum, plan[i].d_Data, plan[i].dataN );
        if ( hipGetLastError() != hipSuccess ) { printf( "reduceKernel() execution failed (GPU %d).\n", i ); exit(EXIT_FAILURE); }
        // Read back GPU results
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N * sizeof( float ), hipMemcpyDeviceToHost, plan[i].stream ) );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    
    // Process GPU results
    printf( "Process GPU results on %d GPUs...\n", GPU_N );
    for( i = 0; i < GPU_N; i++ ) {
        float sum;
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        // Wait for all operations to finish
        hipStreamSynchronize( plan[i].stream );
        // Finalize GPU reduction for current subvector
        sum = 0;
        for( j = 0; j < ACCUM_N; j++ ) {
            sum += plan[i].h_Sum_from_device[j];
        }
        *( plan[i].h_Sum ) = ( float ) sum;
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    double gpuTime = GetTimer();


#ifdef CUPTI_ONLY
    size_t size = 1024;
    size_t sizeBytes = size*sizeof(uint64_t);
    uint64_t buffer[size];
    uint64_t tmp[size];     for (int jj=0; jj<1024; jj++) tmp[jj]=0;
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CU_ERROR( hipCtxSynchronize( ), "hipCtxSynchronize" );
        CHECK_CUPTI_ERROR( cuptiEventGroupReadEvent ( eg[i], CUPTI_EVENT_READ_FLAG_NONE, myevent[i], &sizeBytes, &tmp[0] ), "cuptiEventGroupReadEvent" );
        buffer[i] = tmp[0];
        printf( "CUPTI %s device %d counterValue %u (on one domain, may need to be multiplied by num of domains)\n", cuptiEventName, i, buffer[i] );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
#endif

#ifdef PAPI
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CU_ERROR( hipCtxSynchronize( ), "hipCtxSynchronize" );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    retval = PAPI_stop( EventSet, values );                                         // Stop (will read values).
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_stop failed\n" );
    for( i = 0; i < eventCount; i++ )
        printf( "PAPI counterValue %12lld \t\t --> %s \n", values[i], EventName[i] );

    retval = PAPI_cleanup_eventset( EventSet );
    if( retval != PAPI_OK )  fprintf( stderr, "PAPI_cleanup_eventset failed\n" );
    retval = PAPI_destroy_eventset( &EventSet );
    if( retval != PAPI_OK ) fprintf( stderr, "PAPI_destroy_eventset failed\n" );
    PAPI_shutdown();
#endif

    sumGPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        sumGPU += h_SumGPU[i];
    }
    printf( "  GPU Processing time: %f (ms)\n", gpuTime );

    // Compute on Host CPU
    printf( "Computing the same result with Host CPU...\n" );
    StartTimer();
    sumCPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        for( j = 0; j < plan[i].dataN; j++ ) {
            sumCPU += plan[i].h_Data[j];
        }
    }
    double cpuTime = GetTimer();
    if (gpuTime > 0) {
        printf( "  CPU Processing time: %f (ms) (speedup %.2fX)\n", cpuTime, (cpuTime/gpuTime) );
    } else {
        printf( "  CPU Processing time: %f (ms)\n", cpuTime);
    }

    // Compare GPU and CPU results
    printf( "Comparing GPU and Host CPU results...\n" );
    diff = fabs( sumCPU - sumGPU ) / fabs( sumCPU );
    printf( "  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU );
    printf( "  Relative difference: %E \n", diff );

    // Cleanup and shutdown
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Sum_from_device ) );
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Data ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Sum ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Data ) );
        // Shut down this GPU
        CHECK_CUDA_ERROR( hipStreamDestroy( plan[i].stream ) );
    }

#ifdef CUPTI_ONLY
    free(myevent);
#endif 
    
    exit( ( diff < 1e-5 ) ? EXIT_SUCCESS : EXIT_FAILURE );
}

