#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/** 
 * @file    HelloWorld.c
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:	<your name here>
 *			<your email address>
 * test case for Example component 
 * 
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *	with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *	of the component tests (if all tests are added to a directory named
 *	'tests' in the specific component dir).
 *	See components/README for more details.
 *
 *	The string "Hello World!" is mangled and then restored.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "papi.h"
#include "papi_test.h"

#define NUM_EVENTS 1
#define PAPI 1

// Prototypes
__global__ void helloWorld(char*);


// Host function
int main(int argc, char** argv)
{
#ifdef PAPI
	int retval, i;
	int EventSet = PAPI_NULL;
	long long values[NUM_EVENTS];
	/* REPLACE THE EVENT NAME 'PAPI_FP_OPS' WITH A CUDA EVENT 
	   FOR THE CUDA DEVICE YOU ARE RUNNING ON.
	   RUN papi_native_avail to get a list of CUDA events that are 
	   supported on your machine */
        //char *EventName[] = { "PAPI_FP_OPS" };
        char const *EventName[] = { "cuda:::event:elapsed_cycles_sm:device=0" };
	int events[NUM_EVENTS];
	int eventCount = 0;
	int quiet;

	/* Set TESTS_QUIET variable */
	quiet=tests_quiet( argc, argv );
	
	/* PAPI Initialization */
	retval = PAPI_library_init( PAPI_VER_CURRENT );
	if( retval != PAPI_VER_CURRENT ) {
		if (!quiet) printf("PAPI init failed\n");
		test_fail(__FILE__,__LINE__,
			"PAPI_library_init failed", 0 );
	}

	if (!quiet) {
		printf( "PAPI_VERSION     : %4d %6d %7d\n",
			PAPI_VERSION_MAJOR( PAPI_VERSION ),
			PAPI_VERSION_MINOR( PAPI_VERSION ),
			PAPI_VERSION_REVISION( PAPI_VERSION ) );
	}

	/* convert PAPI native events to PAPI code */
	for( i = 0; i < NUM_EVENTS; i++ ){
                retval = PAPI_event_name_to_code( (char *)EventName[i], &events[i] );
		if( retval != PAPI_OK ) {
			fprintf( stderr, "PAPI_event_name_to_code failed\n" );
			continue;
		}
		eventCount++;
		if (!quiet) printf( "Name %s --- Code: %#x\n", EventName[i], events[i] );
	}

	/* if we did not find any valid events, just report test failed. */
	if (eventCount == 0) {
		if (!quiet) printf( "Test FAILED: no valid events found.\n");
		test_skip(__FILE__,__LINE__,"No events found",0);
		return 1;
	}
	
	retval = PAPI_create_eventset( &EventSet );
	if( retval != PAPI_OK ) {
		if (!quiet) printf( "PAPI_create_eventset failed\n" );
		test_fail(__FILE__,__LINE__,"Cannot create eventset",retval);
	}	

        // If multiple GPUs/contexts were being used, 
        // you need to switch to each device before adding its events
        // e.g. hipSetDevice( 0 );
	retval = PAPI_add_events( EventSet, events, eventCount );
	if( retval != PAPI_OK ) {
		fprintf( stderr, "PAPI_add_events failed\n" );
	}

	retval = PAPI_start( EventSet );
	if( retval != PAPI_OK ) {
		fprintf( stderr, "PAPI_start failed\n" );
	}
#endif


	int j;
	
	// desired output
	char str[] = "Hello World!";

	// mangle contents of output
	// the null character is left intact for simplicity
	for(j = 0; j < 12; j++) {
		str[j] -= j;
		//printf("str=%s\n", str);
	}

	
	// allocate memory on the device
	char *d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);
	
	// copy the string to the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);
	
	// set the grid and block sizes
	dim3 dimGrid(2); // one block per word
	dim3 dimBlock(6); // one thread per character

	// invoke the kernel
	helloWorld<<< dimGrid, dimBlock >>>(d_str);

	// retrieve the results from the device
	hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
	
	// free up the allocated memory on the device
	hipFree(d_str);
	
	if (!quiet) printf("END: %s\n", str);

	
#ifdef PAPI
	retval = PAPI_stop( EventSet, values );
	if( retval != PAPI_OK )
		fprintf( stderr, "PAPI_stop failed\n" );

	retval = PAPI_cleanup_eventset(EventSet);
	if( retval != PAPI_OK )
		fprintf(stderr, "PAPI_cleanup_eventset failed\n");

	retval = PAPI_destroy_eventset(&EventSet);
	if (retval != PAPI_OK)
		fprintf(stderr, "PAPI_destroy_eventset failed\n");

	PAPI_shutdown();

	for( i = 0; i < eventCount; i++ )
		if (!quiet) printf( "%12lld \t\t --> %s \n", values[i], EventName[i] );
#endif

	test_pass(__FILE__);

	return 0;
}


// Device kernel
__global__ void
helloWorld(char* str)
{
	// determine where in the thread grid we are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// unmangle output
	str[idx] += idx;
}

