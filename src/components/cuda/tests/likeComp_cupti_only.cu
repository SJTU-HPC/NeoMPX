#include "hip/hip_runtime.h"
/* 
 * This example is taken from the NVIDIA documentation (Copyright 1993-2013
 * NVIDIA Corporation) and has been adapted to show the use of CUPTI in
 * collecting event counters for multiple GPU contexts.
 *
 * 'likeComp' does the job the component does: breaking the metric events
 * out into a list and then building a group from that list, and trying to 
 * read it.
 */

/*
 * This software contains source code provided by NVIDIA Corporation
 *
 * According to the Nvidia EULA (compute 5.5 version)
 * http://developer.download.nvidia.com/compute/cuda/5_5/rel/docs/EULA.pdf
 *
 * Chapter 2. NVIDIA CORPORATION CUDA SAMPLES END USER LICENSE AGREEMENT
 * 2.1.1. Source Code
 * Developer shall have the right to modify and create derivative works with the Source
 * Code. Developer shall own any derivative works ("Derivatives") it creates to the Source
 * Code, provided that Developer uses the Materials in accordance with the terms and
 * conditions of this Agreement. Developer may distribute the Derivatives, provided that
 * all NVIDIA copyright notices and trademarks are propagated and used properly and
 * the Derivatives include the following statement: “This software contains source code
 * provided by NVIDIA Corporation.”
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cupti.h>
#include <timer.h>

#include "papi.h"
#include "papi_test.h"

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

#include "simpleMultiGPU.h"

// //////////////////////////////////////////////////////////////////////////////
// Data configuration
// //////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
const int DATA_N = 48576 * 32;
char *NameToCollect = NULL;

#define CHECK_CU_ERROR(err, cufunc)                                     \
    if (err != hipSuccess) { printf ("Error %d for CUDA Driver API function '%s'\n", err, cufunc); return -1; }

#define CHECK_CUDA_ERROR(err)                                           \
    if (err != hipSuccess) { printf ("%s:%i Error %d for CUDA [%s]\n", __FILE__, __LINE__, err, hipGetErrorString(err) ); return -1; }

#define CUPTI_CALL(call)                                                      \
do {                                                                          \
    CUptiResult _status = call;                                               \
    if (_status != CUPTI_SUCCESS) {                                           \
        const char *errstr;                                                   \
        cuptiGetResultString(_status, &errstr);                               \
        fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",  \
                __FILE__, __LINE__, #call, errstr);                           \
        exit(-1);                                                             \
    }                                                                         \
} while (0)

#define CHECK_ALLOC_ERROR(var)                                                 \
do {                                                                           \
    if (var == NULL) {                                                         \
        fprintf(stderr, "%s:%d: Error: Memory Allocation Failed \n",           \
                __FILE__, __LINE__);                                           \
        exit(-1);                                                              \
    }                                                                          \
} while (0)

// //////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA SDK sample describing
// reduction optimization strategies
// //////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel( float *d_Result, float *d_Input, int N )
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;
    
    for( int pos = tid; pos < N; pos += threadN )
        sum += d_Input[pos];
    
    d_Result[tid] = sum;
}

static void printUsage() {
    printf("usage: Perform a CUPTI only test of an event or metric.\n");
    printf("       -help           : display help message\n");
    printf("       EVENT_NAME      : or Metric, must be the LAST argument, after any flags.\n");
    printf("Note the PAPI prefix of 'cuda:::event:' or 'cuda:::metric:' should be left off,\n");
    printf("also any ':device=n' suffix. Those are PAPI added elements for disambiguation. \n");
}

void parseCommandLineArgs(int argc, char *argv[])
{
    if (argc < 2) {
        printf("Invalid number of options\n");
        printUsage();
        exit(0);
    }

   NameToCollect = argv[1];                                 // Record name to collect. 
} // end routine.

//-----------------------------------------------------------------------------
// Return a text version with B, KB, MB, GB or TB. 
//-----------------------------------------------------------------------------
#define DIM(x) (sizeof(x)/sizeof(*(x)))
void calculateSize(char *result, uint64_t size)
{
    int i;

    const char *sizes[] = { "TB", "GB", "MB", "KB", "B" };
    uint64_t exbibytes = 1024ULL * 1024ULL * 1024ULL * 1024ULL;

    uint64_t multiplier = exbibytes;

    for(i = 0; (unsigned) i < DIM(sizes); i++, multiplier /= (uint64_t) 1024) {
        if(size < multiplier)
            continue;
        sprintf(result, "%.1f %s", (float) size / multiplier, sizes[i]);
        return;
    }
    strcpy(result, "0");
    return;
} // end routine


//-------------------------------------------------------------------------------------------------
// Returns the values in the event groups. Caller must know the number of events, and eventValues
// must be large enough to hold that many. eventIDArray must be large enough to hold that many 
// event IDs.
//-------------------------------------------------------------------------------------------------
void readEventGroup(CUpti_EventGroup eventGroup,
                    hipDevice_t dev, 
                    uint32_t numEvents,
                    CUpti_EventID *eventIdArray,
                    uint64_t *eventValues) {

    size_t bufferSizeBytes, numCountersRead;
    size_t eventIdArrayBytes= sizeof(CUpti_EventID) * numEvents;
    size_t numTotalInstancesSize = 0;
    uint64_t numTotalInstances = 0;
    uint32_t i = 0, j = 0;
    CUpti_EventDomainID domainId;
    size_t domainSize;

    domainSize = sizeof(CUpti_EventDomainID);

    CUPTI_CALL(cuptiEventGroupGetAttribute(eventGroup, 
                                           CUPTI_EVENT_GROUP_ATTR_EVENT_DOMAIN_ID, 
                                           &domainSize, 
                                           (void *)&domainId));

    numTotalInstancesSize = sizeof(uint64_t);

    CUPTI_CALL(cuptiDeviceGetEventDomainAttribute(dev, 
                                              domainId, 
                                              CUPTI_EVENT_DOMAIN_ATTR_TOTAL_INSTANCE_COUNT, 
                                              &numTotalInstancesSize, 
                                              (void *)&numTotalInstances));

    printf("LINE %i, DeviceEventDomainAttribute numTotalInstances=%llu.\n", __LINE__, numTotalInstances);

    bufferSizeBytes = sizeof(uint64_t) * numEvents * numTotalInstances;
    uint64_t *eventValueArray = (uint64_t *) malloc(bufferSizeBytes);
    CHECK_ALLOC_ERROR(eventValueArray);

    for (i=0; i<numEvents; i++) eventValues[i]=0;                               // init the values.
    
    CUPTI_CALL(cuptiEventGroupReadAllEvents(eventGroup, 
                                            CUPTI_EVENT_READ_FLAG_NONE,
                                            &bufferSizeBytes, 
                                            eventValueArray, 
                                            &eventIdArrayBytes, 
                                            eventIdArray, 
                                            &numCountersRead));

    printf("LINE %i, numCountersRead=%u.\n", __LINE__, numCountersRead);
    if (numCountersRead != numEvents) {
        if (numCountersRead > numEvents) exit(-1);
    }
    
    // Arrangement of 2-d Array returned in eventValueArray:
    //    domain instance 0: event0 event1 ... eventN
    //    domain instance 1: event0 event1 ... eventN
    //    ...
    //    domain instance M: event0 event1 ... eventN
    // But we accumulate by column, event[0], event[1], etc.

    for (i = 0; i < numEvents; i++) {                   // outer loop column traversal.
        for (j = 0; j < numTotalInstances; j++) {       // inner loop row traversal.
            eventValues[i] += eventValueArray[i + numEvents * j];
        }
    }

    free(eventValueArray);                              // Done with this.
} // end routine. 


//-------------------------------------------------------------------------------------------------
// For reading a metric. This still requires a group of events.
// This cannot read a metric that requires more than one group; if you need that, we need to pass
// a set instead, and loop through the groups in the set, and accumulate a table of the collected
// events. TC
//-------------------------------------------------------------------------------------------------
void readMetricValue(CUpti_EventGroup eventGroup, uint32_t numEvents,
                    hipDevice_t dev, CUpti_MetricID *metricId,
                    uint64_t ns_timeDuration,
                    CUpti_MetricValue *metricValue) {
    int i;
    uint64_t *eventValues = NULL;
    CUpti_EventID *eventIDs;

    size_t eventValuesSize = sizeof(uint64_t) * numEvents;
    size_t eventIDsSize = sizeof(CUpti_EventID) * numEvents;

    eventValues = (uint64_t *) malloc(eventValuesSize);
    CHECK_ALLOC_ERROR(eventValues);

    eventIDs = (CUpti_EventID *) malloc(eventIDsSize);
    CHECK_ALLOC_ERROR(eventIDs);

    readEventGroup(eventGroup, dev, numEvents, eventIDs, eventValues);          // Read the event group.
    for (i=0; i<numEvents; i++) {
        printf("   readMetricValue: EventID %lu=read %lu.\n", eventIDs[i], eventValues[i]);
    }

    CUPTI_CALL(cuptiMetricGetValue(dev, metricId[0],
        eventIDsSize, eventIDs, 
        eventValuesSize, eventValues, 
        ns_timeDuration, metricValue));

    free(eventValues);
    free(eventIDs);
} // end routine.


  // Print metric value, we format based on the value kind
int printMetricValue(CUpti_MetricID metricId, CUpti_MetricValue metricValue, 
        const char *metricName) {

    CUpti_MetricValueKind valueKind;
    char str[64];
    size_t valueKindSize = sizeof(valueKind);

    CUPTI_CALL(cuptiMetricGetAttribute(metricId, CUPTI_METRIC_ATTR_VALUE_KIND,
                                       &valueKindSize, &valueKind));
    switch (valueKind) {

    case CUPTI_METRIC_VALUE_KIND_DOUBLE:
        printf("%s = %f\n", metricName, metricValue.metricValueDouble);
        break;

    case CUPTI_METRIC_VALUE_KIND_UINT64:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueUint64);
        printf("%s\n", str);
        break;

    case CUPTI_METRIC_VALUE_KIND_INT64:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueInt64);
        printf("%s\n", str);
        break;

    case CUPTI_METRIC_VALUE_KIND_PERCENT:
        printf("%s = %.2f%%\n", metricName, metricValue.metricValueDouble);
        break;

    case CUPTI_METRIC_VALUE_KIND_THROUGHPUT:
        printf("%s = ", metricName);
        calculateSize(str, (uint64_t)metricValue.metricValueThroughput);
        printf("%s\n", str);
        break;

    default:
        fflush(stdout);
        fprintf(stderr, "error: unknown value kind = %li\n", valueKind);
        return -1;                                                      // indicate failure.
    }

    return 0;                                                           // indicate success.
} // end routine.


// //////////////////////////////////////////////////////////////////////////////
// Program main
// //////////////////////////////////////////////////////////////////////////////
int main( int argc, char **argv )
{
    // Solver config
    TGPUplan plan[MAX_GPU_COUNT];
    // GPU reduction results
    float h_SumGPU[MAX_GPU_COUNT];
    float sumGPU;
    double sumCPU, diff;
    int i, j, gpuBase, GPU_N;
    
    const int BLOCK_N = 32;
    const int THREAD_N = 256;
    const int ACCUM_N = BLOCK_N * THREAD_N;

    hipCtx_t ctx[MAX_GPU_COUNT];
    
    printf( "Starting cudaTest_cupti_only.\n" );

    // Parse command line arguments
    parseCommandLineArgs(argc, argv);
    
    // Report on the available CUDA devices
    int computeCapabilityMajor = 0, computeCapabilityMinor = 0;
    int runtimeVersion = 0, driverVersion = 0;
    char deviceName[64];
    hipDevice_t device[MAX_GPU_COUNT];
    CHECK_CUDA_ERROR( hipGetDeviceCount( &GPU_N ) );
    if( GPU_N > MAX_GPU_COUNT ) GPU_N = MAX_GPU_COUNT;
    printf( "CUDA-capable device count: %i\n", GPU_N );
    for ( i=0; i<GPU_N; i++ ) {
        CHECK_CU_ERROR( hipDeviceGet( &device[i], i ), "hipDeviceGet" );
        CHECK_CU_ERROR( hipDeviceGetName( deviceName, 64, device[i] ), "hipDeviceGetName" );
        CHECK_CU_ERROR( hipDeviceGetAttribute( &computeCapabilityMajor, 
            hipDeviceAttributeComputeCapabilityMajor, device[i]), "hipDeviceGetAttribute");
        CHECK_CU_ERROR( hipDeviceGetAttribute( &computeCapabilityMinor, 
            hipDeviceAttributeComputeCapabilityMinor, device[i]), "hipDeviceGetAttribute");
        hipRuntimeGetVersion( &runtimeVersion );
        hipDriverGetVersion( &driverVersion );
        printf( "CUDA Device %d: %s : computeCapability %d.%d runtimeVersion %d.%d driverVersion %d.%d\n", 
            i, deviceName, computeCapabilityMajor, computeCapabilityMinor, 
            runtimeVersion/1000, (runtimeVersion%100)/10, driverVersion/1000, (driverVersion%100)/10 );
        if ( computeCapabilityMajor < 2 ) {
            printf( "CUDA Device %d compute capability is too low... will not add any more GPUs\n", i );
            GPU_N = i;
            break;
        }
    } // end for each device.

    uint32_t cupti_linked_version;
    cuptiGetVersion( &cupti_linked_version );
    printf("CUPTI version: Compiled against version %d; Linked against version %d\n", 
            CUPTI_API_VERSION, cupti_linked_version );
    
    // create one context per device
    for (i = 0; i < GPU_N; i++) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR( hipCtxCreate( &(ctx[i]), 0, device[i] ), "hipCtxCreate" );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    printf("Searching for '%s'.\n", NameToCollect);
    CUptiResult     myCURes;
    CUpti_EventID   eventId;
    CUpti_MetricID  metricId;
    CUpti_MetricValueKind metricKind;
    size_t          metricKindSize = sizeof(CUpti_MetricValueKind);
    uint32_t        numSubs; // Number of sub-events in Metric.
    
    int isMetric = 0;                                           // Presume this is not a metric.
    int numEventGroups = 0;
    int numMetricEvents[MAX_GPU_COUNT]={0};
    size_t sizeInt = sizeof(int);

    myCURes = cuptiEventGetIdFromName(0, NameToCollect, &eventId);
    if (myCURes == CUPTI_SUCCESS) {
        printf("Found '%s' as an event.\n", NameToCollect);
    } else {
        myCURes = cuptiMetricGetIdFromName(0, NameToCollect, &metricId);
        if (myCURes == CUPTI_SUCCESS) {
            isMetric = 1;                                       // remember we found a metric.
            printf("Found '%s' as a metric.\n", NameToCollect);
        } else {
            printf("'%s' not found, as event or as metric.\n", NameToCollect);
            exit(-1);
        }
    }

    printf( "Generating input data...\n" );
    
    // Subdividing input data across GPUs
    // Get data sizes for each GPU
    for( i = 0; i < GPU_N; i++ )
        plan[i].dataN = DATA_N / GPU_N;
    // Take into account "odd" data sizes
    for( i = 0; i < DATA_N % GPU_N; i++ )
        plan[i].dataN++;
    
    // Assign data ranges to GPUs
    gpuBase = 0;
    for( i = 0; i < GPU_N; i++ ) {
        plan[i].h_Sum = h_SumGPU + i; // point within h_SumGPU array
        gpuBase += plan[i].dataN;
    }

  
    // Create streams for issuing GPU command asynchronously and allocate memory (GPU and System page-locked)
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CUDA_ERROR( hipStreamCreate( &plan[i].stream ) );
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Data, plan[i].dataN * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipMalloc( ( void ** ) &plan[i].d_Sum, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Sum_from_device, ACCUM_N * sizeof( float ) ) );
        CHECK_CUDA_ERROR( hipHostMalloc( ( void ** ) &plan[i].h_Data, plan[i].dataN * sizeof( float ) ) );
        for( j = 0; j < plan[i].dataN; j++ ) {
            plan[i].h_Data[j] = ( float ) rand() / ( float ) RAND_MAX;
        }
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    
    // Create the group(s) needed to read the metric or event.
    CUpti_EventGroup eg[MAX_GPU_COUNT];                                 // event group only.
    CUpti_EventGroupSets* egs[MAX_GPU_COUNT];                           // need event group sets for metric.
    
    if (isMetric) {                                                     // If it is a metric, need a set.
        printf("Setup CUPTI counters internally for metric '%s'.\n", NameToCollect);
        for ( i=0; i<GPU_N; i++ ) {                                         // For every device, 
            CHECK_CUDA_ERROR( hipSetDevice( i ) );
            CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
            CUPTI_CALL(cuptiSetEventCollectionMode(ctx[i], 
                CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS));   // note: CONTINOUS v. KERNEL made no difference in result.


            // Here is where the change occurs. We have metricId.
            // First, get number of events.
            
            CUPTI_CALL(cuptiMetricGetNumEvents(metricId, &numSubs));    // Get number of events needed for metric.
            size_t sizeBytes = numSubs * sizeof(CUpti_EventID);         // bytes needed to store events.
            CUpti_EventID *subEventIds = (CUpti_EventID*) malloc(sizeBytes);        // Get the space for them.
            CUPTI_CALL(cuptiMetricEnumEvents(metricId, &sizeBytes, subEventIds));   // Collect the events.
            
            for (j=0; j<numSubs; j++) printf("Metric subEvent %i: %lu\n", j, subEventIds[j]);
            
            CUPTI_CALL(cuptiMetricGetAttribute(                         // Get the kind.
                metricId, 
                CUPTI_METRIC_ATTR_VALUE_KIND, 
                &metricKindSize, &metricKind));
            printf("Metric value kind = %i.\n", metricKind);
 
            CUPTI_CALL(cuptiEventGroupSetsCreate(                           // create event group sets.
                ctx[i],
                sizeBytes, subEventIds,
                &egs[i]));                              

//          The proper way to do it.
//          CUPTI_CALL(cuptiMetricCreateEventGroupSets(ctx[i], 
//              sizeof(CUpti_MetricID), &metricId, &egs[i]));               // Get the pointer to sets.
            
            printf("Metric device %i requires %i sets.\n", i, egs[i]->numSets);
            if (egs[i]->numSets > 1) {
                printf("'%s' requires multiple application runs to complete. Aborting.\n", NameToCollect);
                exit(-1);
            }

            numEventGroups = egs[i]->sets[0].numEventGroups;                // collect groups in only set.
            if (numEventGroups > 1) {
                printf("'%s' requires multiple groups to complete metric. Aborting.\n", NameToCollect);
                exit(-1);
            }

            // DEBUG note: This has to change to support metrics with multiple
            // groups, if we ever see them.  can't use eg[i], for example,
            // you'd need a different one on each GPU. Tony C.

            for (j=0; j<numEventGroups; j++) {
                uint32_t one = 1;
                eg[i] = egs[i]->sets[0].eventGroups[j];                             // Copy the group.
                CUPTI_CALL(cuptiEventGroupSetAttribute(eg[i], 
                    CUPTI_EVENT_GROUP_ATTR_PROFILE_ALL_DOMAIN_INSTANCES,
                    sizeof(uint32_t), &one));
                CUPTI_CALL(cuptiEventGroupGetAttribute(
                    eg[i], CUPTI_EVENT_GROUP_ATTR_NUM_EVENTS,
                    &sizeInt, &numMetricEvents[i]));                                // read # of events on this device.
                printf("Group %i has %i events.\n", j+1, numMetricEvents[i]);

                size_t subSize = numMetricEvents[i] * sizeof(CUpti_EventID);        // size in bytes.
                CUpti_EventID *subEvents = (CUpti_EventID*) malloc(subSize);
                CUPTI_CALL( cuptiMetricEnumEvents(metricId, &subSize, subEvents));
                int k;
                for (k=0; k<numMetricEvents[i]; k++) {
                    printf("    Group %i event %i ID=%lu\n", j+1, k, subEvents[k]);
                }
    
                free(subEvents);                                                    // free memory used for diagnostic.
            } 

            CUPTI_CALL(cuptiEventGroupSetEnable(&egs[i]->sets[0]));                 // Enable all groups in set.

            CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), 
                "hipCtxPopCurrent" );
        } // end of devices.
    } else {                                                            // If it is an event, just need one group. 
        printf("Setup CUPTI counters internally for event '%s' (CUPTI_ONLY)\n", NameToCollect);

        for ( i=0; i<GPU_N; i++ ) {                                     // For every device, 
            CHECK_CUDA_ERROR( hipSetDevice( i ) );
            CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
            CUPTI_CALL(cuptiSetEventCollectionMode(ctx[i], 
                CUPTI_EVENT_COLLECTION_MODE_CONTINUOUS));
            CUPTI_CALL( cuptiEventGroupCreate( ctx[i], &eg[i], 0 ));
            CUPTI_CALL( cuptiEventGroupAddEvent(eg[i], eventId));
            CUPTI_CALL( cuptiEventGroupEnable( eg[i] )); 
            CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), 
                "hipCtxPopCurrent" );
        } // end of devices.
    } // end of if metric/event.
    
    // Start timing and compute on GPU(s)
    printf( "Computing with %d GPUs...\n", GPU_N );
   
    uint64_t ns_timeDuration;                                                   // cuda device time elapsed. 
    uint64_t startTimestamp, endTimestamp;
    CUPTI_CALL(cuptiGetTimestamp(&startTimestamp));                             // We need time in ns for metrics.

    // Copy data to GPU, launch the kernel and copy data back. All asynchronously
    for (i = 0; i < GPU_N; i++) {
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ));
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        // Copy input data from CPU
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].d_Data, plan[i].h_Data, plan[i].dataN * sizeof( float ), hipMemcpyHostToDevice, plan[i].stream ) );
        // Perform GPU computations
        reduceKernel <<< BLOCK_N, THREAD_N, 0, plan[i].stream >>> ( plan[i].d_Sum, plan[i].d_Data, plan[i].dataN );
        if ( hipGetLastError() != hipSuccess ) { printf( "reduceKernel() execution failed (GPU %d).\n", i ); exit(EXIT_FAILURE); }
        // Read back GPU results
        CHECK_CUDA_ERROR( hipMemcpyAsync( plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N * sizeof( float ), hipMemcpyDeviceToHost, plan[i].stream ) );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }
    
    // Process GPU results
    printf( "Process GPU results on %d GPUs...\n", GPU_N );
    for( i = 0; i < GPU_N; i++ ) {
        float sum;
        // Set device
        CHECK_CUDA_ERROR( hipSetDevice( i ) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        // Wait for all operations to finish
        hipStreamSynchronize( plan[i].stream );
        // Finalize GPU reduction for current subvector
        sum = 0;
        for( j = 0; j < ACCUM_N; j++ ) {
            sum += plan[i].h_Sum_from_device[j];
        }
        *( plan[i].h_Sum ) = ( float ) sum;
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    CUPTI_CALL(cuptiGetTimestamp(&endTimestamp));
    ns_timeDuration = endTimestamp - startTimestamp;

    double gpuTime = (ns_timeDuration/((double) 1000000.0));                    // convert to ms.

    // Now, we must read the metric/event. 
    size_t size = 1024;
    uint64_t buffer[size];

    for ( i=0; i<GPU_N; i++ ) {                                                 // for each device,
        CHECK_CUDA_ERROR( hipSetDevice( i ) );                                 // point at it.
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CU_ERROR( hipCtxSynchronize( ), "hipCtxSynchronize" );              // wait for all to finish.

        if (isMetric) {                                                         // If we have a metric,
            CUpti_MetricValue metricValue;
            readMetricValue(eg[i], numMetricEvents[i], 
            device[i], &metricId,
            ns_timeDuration, &metricValue);
            printf("Device %i, Metric: ",i);                                    // prefix the printing...
            printMetricValue(metricId, metricValue, NameToCollect);             // Print "name = value\n".
        } else {                                                                // If we have just an event.
            readEventGroup(eg[i], device[i], 
                1, &eventId,                                                    // just 1 event.
                &buffer[i]);
            printf( "CUPTI %s device %d counterValue %u (on one domain, "
                    "may need to be multiplied by num of domains)\n", 
                    NameToCollect, i, buffer[i] );
        }

        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    sumGPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        sumGPU += h_SumGPU[i];
    }
    printf( "  GPU Processing time: %f (ms)\n", gpuTime );

    // Compute on Host CPU
    printf( "Computing the same result with Host CPU...\n" );
    StartTimer();
    sumCPU = 0;
    for( i = 0; i < GPU_N; i++ ) {
        for( j = 0; j < plan[i].dataN; j++ ) {
            sumCPU += plan[i].h_Data[j];
        }
    }
    double cpuTime = GetTimer();
    if (gpuTime > 0) {
        printf( "  CPU Processing time: %f (ms) (speedup %.2fX)\n", cpuTime, (cpuTime/gpuTime) );
    } else {
        printf( "  CPU Processing time: %f (ms)\n", cpuTime);
    }

    // Compare GPU and CPU results
    printf( "Comparing GPU and Host CPU results...\n" );
    diff = fabs( sumCPU - sumGPU ) / fabs( sumCPU );
    printf( "  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU );
    printf( "  Relative difference: %E \n", diff );

    // Cleanup and shutdown
    for( i = 0; i < GPU_N; i++ ) {
        CHECK_CUDA_ERROR( hipSetDevice(i) );
        CHECK_CU_ERROR(hipCtxPushCurrent(ctx[i]), "hipCtxPushCurrent");
        CHECK_CUDA_ERROR( hipStreamSynchronize(plan[i].stream) ); 
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Sum_from_device ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Sum ) );
        CHECK_CUDA_ERROR( hipFree( plan[i].d_Data ) );
        CHECK_CUDA_ERROR( hipStreamDestroy( plan[i].stream ) );
        CHECK_CUDA_ERROR( hipHostFree( plan[i].h_Data ) );
        CHECK_CU_ERROR( hipCtxPopCurrent(&(ctx[i])), "hipCtxPopCurrent" );
    }

    exit( ( diff < 1e-5 ) ? EXIT_SUCCESS : EXIT_FAILURE );
}

