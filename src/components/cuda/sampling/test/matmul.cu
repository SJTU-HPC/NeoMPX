//This is a matrix multiplication program in CUDA without any optimizations
//like tiling, using shared memory etc

#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<assert.h>


__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int width)
{

	//2D thread ID 
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tdx=threadIdx.x;
	int tdy=threadIdx.y;

	int tx=bx*blockDim.x+tdx;
	int ty=by*blockDim.y+tdy;
	
	//Pvalue stores the Pd element that is computed by the thread
	float Pvalue=0;
	for(int k=0;k<width;++k){
		float Mdelement=Md[ty*width+k];
		float Ndelement=Nd[k*width+tx];
		Pvalue += Mdelement*Ndelement;
	}
	//Write the matrix to device memory each thread writes one element
	Pd[ty*width+tx]=Pvalue;
}


int main(int argc, char** argv){

	int width;
	int BlockDim;
	int GridDim;

	if (argc == 3){
		width=atoi(argv[1]);
		BlockDim=atoi(argv[2]);
		GridDim=width/BlockDim;
		printf("Using matrix dimension %dx%d ,Block Dim %dx%d threads per block, Grid Dim %dx%d blocks per grid\n",width,width,BlockDim,BlockDim,GridDim,GridDim);
	}else{
		width=512;
		BlockDim=16;
		GridDim=width/BlockDim;
		printf("Using Default Parameters: matrix dimension %dx%d ,Block Dim %dx%d threads per block, Grid Dim %dx%d blocks per grid\n",width,width,BlockDim,BlockDim,GridDim,GridDim);		
	}
	dim3 dimBlock(BlockDim,BlockDim);
	dim3 dimGrid(GridDim,GridDim);
	hipError_t error;
	hipDeviceProp_t deviceProp;
	int devID=0;
	error=hipGetDevice(&devID);
	if (error != hipSuccess)
	{
		printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
	}

	error=hipGetDeviceProperties(&deviceProp,devID);
	if (error != hipSuccess){
		printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	}else{
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
	}

	int size=width*width*sizeof(float);
	float* M=(float*)malloc(size);
	float* N=(float*)malloc(size);
	float* P=(float*)malloc(size);

	float* Md,*Nd,*Pd;

	if(!(M&&N)){
		printf("Malloc failed\n");
		exit(-1);
	}

	 // initialization of host data
	for (int j = 0; j < width; j++) {
		for (int i = 0; i < width; i++) {
			M[j*width + i] = (float)(rand()%50);
			N[j*width + i] = (float)(rand()%50);
			P[j*width + i] = 0;
		}
	}

	error=hipMalloc((void**)&Md,size);
	if(error!=hipSuccess){
		printf("Device memory allocation for M failed \n");
		exit(-1);
	}
	error=hipMalloc((void**)&Nd,size);
	if(error!=hipSuccess){
		printf("Device memory allocation for N failed \n");
		exit(-1);
	}
	error=hipMalloc((void**)&Pd,size);
	if(error!=hipSuccess){
		printf("Device memory allocation for P failed \n");
		exit(-1);
	}

	error=hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
	if(error!=hipSuccess){
		printf("Device memory copy for M failed \n");
		exit(-1);
	}
	
	error=hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
	if(error!=hipSuccess){
		printf("Device memory copy for N failed \n");
		exit(-1);
	}

	
	hipEvent_t start;
	error=hipEventCreate(&start);
	if(error!=hipSuccess){
		printf("cuda event start failed \n");
		exit(-1);
	}

	hipEvent_t stop;
	error=hipEventCreate(&stop);
	if(error!=hipSuccess){
		printf("cuda event stop failed \n");
		exit(-1);
	}

	error =hipEventRecord(start,NULL);
	if(error!=hipSuccess){
		printf("cuda event start record failed \n");
		exit(-1);
	}
	
	
	MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,width);

//	error=cudaDeviceSynchronize();
	error =hipEventRecord(stop,NULL);
	if(error!=hipSuccess){
		printf("cuda event stop record failed with error=%s\n",hipGetErrorString(error));
		exit(-1);
	}

	error = hipEventSynchronize(stop);
	if(error!=hipSuccess){
		printf("cuda event sync failed :%s\n",hipGetErrorString(error));
		exit(-1);
	}
	


	float msecTotal=0.0f;
	error = hipEventElapsedTime(&msecTotal,start,stop);
	if(error!=hipSuccess){
		printf("cuda elapsed time calculation failed \n");
		exit(-1);
	}

	float msecPerMatrixMul = msecTotal;
	double flopsPerMatrixMul = 2*width*width*width;
	double gigaFlops=(flopsPerMatrixMul*1.0e-9f)/(msecPerMatrixMul/1000.0f);
	printf("Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		    gigaFlops,
			msecPerMatrixMul,
			flopsPerMatrixMul,
			width * width);



	error=hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);
	if(error!=hipSuccess){
		printf("Device memoory copy back for Pd failed \n");
		exit(-1);
	}

	printf("Very slow Host Matrix Mult \n");
	float temp;
	// initialization of host data
	for (int i = 0; i < width; ++i) {
		for ( int j = 0; j < width; ++j) {
			temp=0;
			for(int k=0; k<width; ++k)
				temp+=M[i*width+k]*N[k*width+j];
			if(temp != P[i*width+j]){
				printf("Matrix Mult Screwed Up!! differ in values CPU:%f and GPU:%f \n",temp,P[i*width+j]);
				exit(-1);
			}
		}
		
	}
	
	
	free(M);
	free(N);
	free(P);
	hipFree(Md);	
	hipFree(Nd);	
	hipFree(Pd);	
	hipDeviceReset();
	return 1;

}
